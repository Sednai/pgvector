#include "hip/hip_runtime.h"
#include "ivfgpu.h"
#include <thrust/sort.h>
#include <thrust/device_ptr.h>

#define THREADS_PER_BLOCK 1024

__global__ void calc_euclidean_distances_v0(float* M, float* V, float* C, int N, int L) {
    int row = blockIdx.x*blockDim.x+threadIdx.x;

    if(row < N) {
        float tmp = 0;
        float tmp2;
        for(int i = 0; i < L; i++) {
            tmp2 = M[L*row+i] - V[i];
            tmp += tmp2*tmp2;
        }
        C[row] = sqrt(tmp);
    }
}

__global__ void calc_euclidean_distances_v1(float* M, float* V, float* C, int N, int L) {
    unsigned int indexx = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int stridex = blockDim.x*gridDim.x;
    unsigned int k;

    for(k=indexx; k < N; k += stridex) {
        float tmp = (M[L*k] - V[0])*(M[L*k] - V[0]);
        for(int i = 1; i < L; i++) {
            tmp += (M[L*k+i] - V[i])*(M[L*k+i] - V[i]);
        }
        C[k] = sqrt(tmp);
    }
}

__global__ void calc_squared_euclidean_distances_v0(float* M, float* V, float* C, int N, int L) {
    unsigned int indexx = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int stridex = blockDim.x*gridDim.x;
    unsigned int k;

    for(k=indexx; k < N; k += stridex) {
        float tmp = (M[L*k] - V[0])*(M[L*k] - V[0]);
        for(int i = 1; i < L; i++) {
            tmp += (M[L*k+i] - V[i])*(M[L*k+i] - V[i]);
        }
        C[k] = tmp;
    }
}

__global__ void calc_squared_euclidean_distances_v0b(float* M, float* V, float V2s, float* C, int N, int L) {
    unsigned int indexx = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int stridex = blockDim.x*gridDim.x;
    unsigned int k;

    for(k=indexx; k < N; k += stridex) {
        float tmp = V2s + M[L*k]*M[L*k] - 2*V[0]*M[L*k];
        for(int i = 1; i < L; i++) {
            tmp += M[L*k+i]*M[L*k+i] - 2*V[i]*M[L*k+i];
        }
        C[k] = tmp;
    }
}


__global__ void calc_squared_euclidean_distances_v0c(float* M, float* V, float* C, int N, int L) {
    unsigned int indexx = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int stridex = blockDim.x*gridDim.x;
    unsigned int k;

    __shared__ float VL[THREADS_PER_BLOCK];
    if(threadIdx.x < L)
        VL[threadIdx.x] = V[threadIdx.x];
    
    __syncthreads();
    
    for(k=indexx; k < N; k += stridex) {
        float tmp = (M[L*k] - VL[0])*(M[L*k] - VL[0]);
        for(int i = 1; i < L; i++) {
            tmp += (M[L*k+i] - VL[i])*(M[L*k+i] - VL[i]);
        }
        C[k] = tmp;
    }
}

__global__ void calc_squared_euclidean_distances_wsfilter_v0(float* M, float* V, sort_item* C, const float f, int* p, int N, int L, int probe) {
    unsigned int indexx = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int stridex = blockDim.x*gridDim.x;
    unsigned int k;

    __shared__ float VL[THREADS_PER_BLOCK];
    if(threadIdx.x < L)
        VL[threadIdx.x] = V[threadIdx.x];
    
    __syncthreads();
    
    for(k=indexx; k < N; k += stridex) {
        float tmp = (M[L*k] - VL[0])*(M[L*k] - VL[0]);
        for(int i = 1; i < L; i++) {
            tmp += (M[L*k+i] - VL[i])*(M[L*k+i] - VL[i]);
        }
        if(tmp < f) {
            int pos = atomicAdd(p,1);
            C[pos].distance = tmp;
            C[pos].probe = probe;
            C[pos].pos = k;
        }
    }
}



__global__ void calc_squared_euclidean_distances_v1(float* M, float* V, float* C, int N, int L) {
    unsigned int indexx = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int stridex = blockDim.x*gridDim.x;
    
    unsigned int indexy = blockIdx.y*blockDim.y + threadIdx.y;
    unsigned int stridey = blockDim.y*gridDim.y;

    unsigned int x,y;

    for(x = indexx; x < N; x += stridex) {
        float tmp = 0;
        for(y = indexy; y < L; y += stridey) {
            tmp += (M[L*x+y] - V[y]) * (M[L*x+y] - V[y]);
        }

        atomicAdd(&C[x],tmp);
    }
}

__global__ void calc_squared_euclidean_distances_v2(float* M, float* V, float* C, int N, int L) {
    unsigned int indexx = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int stridex = blockDim.x*gridDim.x;
    
    unsigned int indexy = blockIdx.y*blockDim.y + threadIdx.y;
    unsigned int stridey = blockDim.y*gridDim.y;

    unsigned int x,y;
    
    __shared__ float tmp[32]; 
    
    for(x = indexx; x < N; x += stridex) {
        tmp[threadIdx.x] = 0;
        __syncthreads();
        
        float tmp2 = 0;
        for(y = indexy; y < L; y += stridey) {
            tmp2 += (M[L*x+y] - V[y]) * (M[L*x+y] - V[y]);
        }
        
        atomicAdd(&tmp[threadIdx.x], tmp2);
        
        __syncthreads();
        
        if(threadIdx.y < 1)
            atomicAdd(&C[x],tmp[threadIdx.x]);
    }
}

__global__ void apply_sqrt(float* C, int N) {
    unsigned int indexx = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int stridex = blockDim.x*gridDim.x;

    for(int x = indexx; x < N; x += stridex) {
        C[x] = sqrt(C[x]);
    }
}

__global__ void apply_sqrt_with_seq_filter(float* C, float filter, int N) {
    unsigned int indexx = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int stridex = blockDim.x*gridDim.x;

    for(int x = indexx; x < N; x += stridex) {
        float tmp = (C[x]);
        if( tmp <= filter) 
            C[x] = tmp;
        else
            C[x] = -1;
    }
}

__global__ void nullify(float* C, int N) {
    unsigned int indexx = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int stridex = blockDim.x*gridDim.x;

    for(int x = indexx; x < N; x += stridex) {
        C[x] = 0;
    }
}

void* init_shared_gpu_memory(int size) {
    // Initialize unified memory
    void* M;
    hipMallocManaged(&M, size);
    
    return M;
}

void init_gpu_memory(void** P, int size) {
    // Initialize non-unified memory
    hipMalloc(P, size);    
}

void free_gpu_memory(void* P) {
    hipFree(P);
}

void prefetch_gpu_memory(void* P, int size, int device) {
    hipMemPrefetchAsync(P, size, device);
}

void advise_memory_readonly(void* P, int size, int device ) {
       hipMemAdvise(P, size, hipMemAdviseSetReadMostly, device);
}

void copy_memory_to_gpu(void* T, void* F, int size) {
    hipMemcpy(T, F, size, hipMemcpyHostToDevice);
}

void copy_memory_async_to_gpu(void* T, void* F, int size) {
    hipMemcpyAsync(T, F, size, hipMemcpyHostToDevice);
}

void copy_memory_to_cpu(void* T, void* F, int size) {
    hipMemcpy(T, F, size, hipMemcpyDeviceToHost);
}

void copy_memory_async_to_cpu(void* T, void* F, int size) {
    hipMemcpyAsync(T, F, size, hipMemcpyDeviceToHost);
}

void synchronize_gpu() {
    hipDeviceSynchronize();
}

void calc_distances_gpu_euclidean(float* M, float* V, float* C, int N, int L) {
    
    //calc_euclidean_distances_v0<<<(N+THREADS_PER_BLOCK+1)/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(M, V, C, N, L);    
    //calc_euclidean_distances_v1<<<1024,THREADS_PER_BLOCK>>>(M, V, C, N, L);    
    //calc_euclidean_distances_v1<<<(N+THREADS_PER_BLOCK+1)/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(M, V, C, N, L);   
    
    dim3 DimGrid(1024, 2); 
    dim3 DimBlock(32, 32); 
    
    nullify<<<(N+THREADS_PER_BLOCK+1)/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(C, N);    
    calc_squared_euclidean_distances_v2<<<DimGrid,DimBlock>>>(M, V, C, N, L);    
    apply_sqrt<<<(N+THREADS_PER_BLOCK+1)/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(C, N);    
    
    hipMemPrefetchAsync(C, N*sizeof(float), hipCpuDeviceId);
    hipDeviceSynchronize();
   
}


void calc_squared_distances_gpu_euclidean_nosharedmem(float* M, float* V, float* C, int N, int L) {
    
    calc_squared_euclidean_distances_v0c<<<(N-1+THREADS_PER_BLOCK)/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(M, V, C, N, L);    
}

/*
    Calc euclidean distances and apply < filter
*/
void calc_squared_distances_gpu_euclidean_wsfilter(float* M, float* V, sort_item* C, const float f, int* p, int N, int L, int probe) {

    // Calc distance + filter
    calc_squared_euclidean_distances_wsfilter_v0<<<(N-1+THREADS_PER_BLOCK)/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(M, V, C, f, p, N, L, probe);
}



void calc_squared_distances_gpu_euclidean(float* M, float* V, float* C, int N, int L) {
    
    calc_squared_euclidean_distances_v0<<<(N+THREADS_PER_BLOCK+1)/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(M, V, C, N, L);    
    /*
    dim3 DimGrid(1024, 2); 
    dim3 DimBlock(32, 32); 
    
    nullify<<<(N+THREADS_PER_BLOCK+1)/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(C, N);    
    calc_squared_euclidean_distances_v2<<<DimGrid,DimBlock>>>(M, V, C, N, L);    
    */
    hipMemPrefetchAsync(C, N*sizeof(float), hipCpuDeviceId);
    hipDeviceSynchronize();
}

void calc_squared_distances_gpu_euclidean_mod(float* M, float* V, float V2s, float* C, int N, int L) {
    
    calc_squared_euclidean_distances_v0b<<<(N+THREADS_PER_BLOCK+1)/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(M, V, V2s, C, N, L);    
    /*
    dim3 DimGrid(1024, 2); 
    dim3 DimBlock(32, 32); 
    
    nullify<<<(N+THREADS_PER_BLOCK+1)/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(C, N);    
    calc_squared_euclidean_distances_v2<<<DimGrid,DimBlock>>>(M, V, C, N, L);    
    */
    hipMemPrefetchAsync(C, N*sizeof(float), hipCpuDeviceId);
    hipDeviceSynchronize();
}


struct cmp : public thrust::less<page_item>
{
   __inline__
   __host__ __device__
   bool operator()(const page_item& a, const page_item& b) const {
      return a.distance < b.distance;
   }
};

void sort_array_gpu(page_item* P, int N) {
    thrust::sort(thrust::device, P, P + N, cmp() );
    hipMemPrefetchAsync(P, N*sizeof(page_item), hipCpuDeviceId);
    hipDeviceSynchronize();
}


struct cmp_item : public thrust::less<sort_item>
{
   __inline__
   __host__ __device__
   bool operator()(const sort_item& a, const sort_item& b) const {
      return a.distance < b.distance;
   }
};

void sort_item_array_gpu(sort_item* P, int N) {
    thrust::sort(thrust::device, P, P + N, cmp_item() );
}