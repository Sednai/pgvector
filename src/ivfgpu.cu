#include "hip/hip_runtime.h"
#include <stdio.h>

#define THREADS_PER_BLOCK 1024

__global__ void calc_euclidean_distances_v0(float* M, float* V, float* C, int N, int L) {
    int row = blockIdx.x*blockDim.x+threadIdx.x;

    if(row < N) {
        float tmp = 0;
        float tmp2;
        for(int i = 0; i < L; i++) {
            tmp2 = M[L*row+i] - V[i];
            tmp += tmp2*tmp2;
        }
        C[row] = sqrt(tmp);
    }
}

__global__ void calc_euclidean_distances_v1(float* M, float* V, float* C, int N, int L) {
    unsigned int indexx = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int stridex = blockDim.x*gridDim.x;
    unsigned int k;

    for(k=indexx; k < N; k += stridex) {
        float tmp = (M[L*k] - V[0])*(M[L*k] - V[0]);
        for(int i = 1; i < L; i++) {
            tmp += (M[L*k+i] - V[i])*(M[L*k+i] - V[i]);
        }
        C[k] = sqrt(tmp);
    }
}

__global__ void calc_squared_euclidean_distances_v0(float* M, float* V, float* C, int N, int L) {
    unsigned int indexx = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int stridex = blockDim.x*gridDim.x;
    
    unsigned int indexy = blockIdx.y*blockDim.y + threadIdx.y;
    unsigned int stridey = blockDim.y*gridDim.y;

    unsigned int x,y;

    for(x = indexx; x < N; x += stridex) {
        float tmp = (M[L*x+indexy] - V[indexy]) * (M[L*x+indexy] - V[indexy]);
        
        for(y = indexy+stridey; y < L; y += stridey) {
            tmp += (M[L*x+y] - V[y]) * (M[L*x+y] - V[y]);
        }

        atomicAdd(&C[x],tmp);
    }
}

__global__ void apply_sqrt(float* C, int N) {
    unsigned int indexx = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int stridex = blockDim.x*gridDim.x;

    for(int x = indexx; x < N; x += stridex) {
        C[x] = sqrt(C[x]);
    }
}

__global__ void nullify(float* C, int N) {
    unsigned int indexx = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int stridex = blockDim.x*gridDim.x;

    for(int x = indexx; x < N; x += stridex) {
        C[x] = 0;
    }
}

extern "C" void* init_shared_gpu_memory(int size) {
    // Initialize unified memory
    void* M;
    hipMallocManaged(&M, size);
    
    return M;
}

extern "C" void free_shared_gpu_memory(void* P) {
    hipFree(P);
}

extern "C" void prefetch_gpu_memory(void* P, int size, int device) {
    hipMemPrefetchAsync(P, size, device);
}

extern "C" void advise_memory_readonly(void* P, int size, int device ) {
       hipMemAdvise(P, size, hipMemAdviseSetReadMostly, device);
}

extern "C" void calc_distances_gpu_euclidean(float* M, float* V, float* C, int N, int L) {
       
    //calc_euclidean_distances_v0<<<(N+THREADS_PER_BLOCK+1)/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(M, V, C, N, L);    
    //calc_euclidean_distances_v1<<<1024,THREADS_PER_BLOCK>>>(M, V, C, N, L);    
    calc_euclidean_distances_v1<<<(N+THREADS_PER_BLOCK+1)/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(M, V, C, N, L);   

    /*
    dim3 DimGrid(1024, 4); 
    dim3 DimBlock(128, 8); 
   
    nullify<<<(N+THREADS_PER_BLOCK+1)/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(C, N);    
    calc_squared_euclidean_distances_v0<<<DimGrid,DimBlock>>>(M, V, C, N, L);    
    apply_sqrt<<<(N+THREADS_PER_BLOCK+1)/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(C, N);    
    */
   
    hipMemPrefetchAsync(C, N, hipCpuDeviceId);
    hipDeviceSynchronize();
}

